#include <stdio.h>
#include <hip/hip_runtime.h>

// Error-checking macro
#define cudaCheckError(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line)
{
    if (code != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s %s %d\n",
                hipGetErrorString(code), file, line);
        exit(code);
    }
}

__global__ void add(int a, int b, int *c) {
    *c = a + b;
}

int main(void) {
    int c = 0;
    int *dev_c;

    // Allocate memory on GPU
    cudaCheckError(hipMalloc((void **)&dev_c, sizeof(int)));

    // Launch kernel
    add<<<1, 1>>>(2, 7, dev_c);

    // Check for kernel launch errors
    cudaCheckError(hipGetLastError());
    cudaCheckError(hipDeviceSynchronize());

    // Copy result back
    cudaCheckError(hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost));

    printf("2 + 7 = %d\n", c);

    cudaCheckError(hipFree(dev_c));
    return 0;
}


