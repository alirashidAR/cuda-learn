#include <stdio.h>
#include <hip/hip_runtime.h>

int main() {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);  // Get properties of device 0

    printf("Maximum threads per block: %d\n", prop.maxThreadsPerBlock);
    printf("Maximum block dimensions: (%d, %d, %d)\n",
           prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
    printf("Maximum grid dimensions: (%d, %d, %d)\n",
           prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);

    return 0;
}

